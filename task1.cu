
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4
// 
__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
}


int main(){

  int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

  //Allocate memories for device copies of A,B,C
  
  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));
  
  //Allocate space for host copies of A,B,C and setup input values

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  //Launch MatAdd() kernel on GPU
  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);
  
  //copy results back to host

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

  //cleanup	
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}
