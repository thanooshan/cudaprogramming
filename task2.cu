
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 22

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
}

void randmatfunc(int newmat[N][N]){
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
       
    } 
  printf("\n--------------------------------------\n"); 
}

int main(){

int A[N][N];  
randmatfunc(A);
  
int B[N][N];  
randmatfunc(B);  



  int C[N][N];

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

  //Allocate memories for device copies of A,B,C
  
  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));
  
  //Alloc space for host copies of A,B,C and setup input values

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  
  //Launch MatAdd() kernel on GPU 
  
  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

  //copy results back to host
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

  //cleanup
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

